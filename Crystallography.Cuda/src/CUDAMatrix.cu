#include "hip/hip_runtime.h"
//
// Cardiff University | Computer Science
// Module:     CM3203 One Semester Project (40 Credits)
// Title:      Parallelisation of Matrix Exponentials in C++/CUDA for Quantum Control
// Date:       2016
//
// Author:     Peter Davison
// Supervisor: Dr. Frank C Langbein
// Moderator:  Dr. Irena Spasic
//

// Include header file
#include "CUDAMatrix.cuh"

// KERNELS

__global__ void cudaAdd(thrust::complex<double>* A, thrust::complex<double>* B, thrust::complex<double>* R, int n) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n && col < n) {
		R[row * n + col] = A[row * n + col] + B[row * n + col];
	}
	__syncthreads();
}

__global__ void cudaAddScalar(thrust::complex<double>* A, thrust::complex<double> scalar, thrust::complex<double>* R, int n) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n && col < n) {
		R[row * n + col] = A[row * n + col] + scalar;
	}
	__syncthreads();
}

__global__ void cudaSub(thrust::complex<double>* A, thrust::complex<double>* B, thrust::complex<double>* R, int n) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n && col < n) {
		R[row * n + col] = A[row * n + col] - B[row * n + col];
	}
	__syncthreads();
}

__global__ void cudaSubScalar(thrust::complex<double>* A, thrust::complex<double> scalar, thrust::complex<double>* R, int n) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n && col < n) {
		R[row * n + col] = A[row * n + col] - scalar;
	}
	__syncthreads();
}

__global__ void cudaMul(thrust::complex<double>* A, thrust::complex<double>* B, thrust::complex<double>* R, int n) {
	thrust::complex<double> sum = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n && col < n) {
		for (int i = 0; i < n; i++) {
			sum += A[row * n + i] * B[i * n + col];
		}
	}
	R[row * n + col] = sum;
	__syncthreads();
}

__global__ void cudaMulScalar(thrust::complex<double>* A, thrust::complex<double> scalar, thrust::complex<double>* R, int n) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n && col < n) {
		R[row * n + col] = A[row * n + col]  * scalar;
	}
	__syncthreads();
}

__global__ void cudaAbs(thrust::complex<double>* A, thrust::complex<double>* R, int n) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n && col < n) {
		R[row * n + col] = abs(A[row * n + col]);
	}
	__syncthreads();
}

// MEMORY HANDLERS

void CUDAMatrix::alloc() {
	h_matrix = (std::complex<double>*) malloc(size);
	hipError_t result = hipMalloc((void**) &d_matrix, size);
	if (result != hipSuccess) {
		throw std::runtime_error("Failed to allocate device memory");
	}
}

void CUDAMatrix::dealloc() {
	free(h_matrix);
	hipError_t result = hipFree(d_matrix);
	if (result != hipSuccess) {
		throw std::runtime_error("Failed to free device memory");
	}
}
 
// CUDA STUFF

void CUDAMatrix::syncHost() {
	if (isInitialised()) {
		hipError_t result = hipMemcpy(h_matrix, d_matrix, size, hipMemcpyDeviceToHost);
		if (result != hipSuccess) {
			throw std::runtime_error("Failed to allocate device memory");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

void CUDAMatrix::syncDevice() {
	if (isInitialised()) {
		hipError_t result = hipMemcpy(d_matrix, h_matrix, size, hipMemcpyHostToDevice);
		if (result != hipSuccess) {
			throw std::runtime_error("Failed to allocate device memory");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDAMatrix::cudaParams CUDAMatrix::getCUDAParams(int rows, int cols) {
	cudaParams cp;
	cp.tpb = dim3(rows, cols);
	cp.bpg = dim3(1, 1);
	
	//�ȉ����R�����g�A�E�g���Ă݂�
	if (rows*cols > 512*512) {
		cp.tpb.x = 512;
		cp.tpb.y = 512;
		cp.bpg.x = (int) (ceil(double(rows) / double(cp.tpb.x)));
		cp.bpg.y = (int) (ceil(double(cols) / double(cp.tpb.y)));
	}
	return cp;
}

// INTERNAL PADE APPROXIMATION CODE

int CUDAMatrix::ell(CUDAMatrix& A, double coef, int m) {
	CUDAMatrix sA(A.getNumRows());
	CUDAMatrix::abs(A, sA);
	double scale = std::pow(coef, (1 / (double) (2 * m + 1)));
	CUDAMatrix::mul(sA, scale, sA);
	//double alpha = sA.getNormAm(2 * m + 1) / A.getNorm(1);     2 LINES BELOW ARE TEMPORARY REPLACEMENT
	CUDAMatrix::pow(sA, (2 * m + 1), sA);
	double alpha = sA.getNorm(1) / (double) (A.getNorm(1));
	/////
	return utils::max((int) (ceil(log2(2 * alpha / std::numeric_limits<double>::epsilon()) / (2 * m))), 0);
}

CUDAMatrix::padeParams CUDAMatrix::getPadeParams(CUDAMatrix& A) {
	// Init
	double d4, d6, d8, d10, eta1, eta3, eta4, eta5;
	int ar = A.getNumRows();
	int ac = A.getNumCols();
	std::vector<double> theta;
	std::vector<double> coef;
	// Init P;
	padeParams p;
	p.pow.resize(11);
	p.scale = 0;
	// Get coefficients and theta values
	coef = {
		(1 / 100800.0),
		(1 / 10059033600.0),
		(1 / 4487938430976000.0),
		(1 / 5914384781877411840000.0),
		(1 / 113250775606021113483283660800000000.0)
	};
	theta = {
		1.495585217958292e-002,
		2.539398330063230e-001,
		9.504178996162932e-001,
		2.097847961257068e+000,
		5.371920351148152e+000
	};
	// Get powers of A
	p.pow[2] = new CUDAMatrix(ar, ac);
	p.pow[4] = new CUDAMatrix(ar, ac);
	p.pow[6] = new CUDAMatrix(ar, ac);
	p.pow[8] = new CUDAMatrix(ar, ac);
	p.pow[10] = new CUDAMatrix(ar, ac);
	cudaParams cp = getCUDAParams(A.getNumRows(), A.getNumCols());
	cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, A.d_matrix, p.pow[2]->d_matrix, ar);
	cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (p.pow[2]->d_matrix, p.pow[2]->d_matrix, p.pow[4]->d_matrix, ar);
	cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (p.pow[2]->d_matrix, p.pow[4]->d_matrix, p.pow[6]->d_matrix, ar);
	cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (p.pow[4]->d_matrix, p.pow[4]->d_matrix, p.pow[8]->d_matrix, ar);
	cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (p.pow[4]->d_matrix, p.pow[6]->d_matrix, p.pow[10]->d_matrix, ar);

	// NOT IDEAL .. PERFORM GETNORM ON DEVICE IF POSSIBLE. THIS MEANS SYNCING BETWEEN HOST AND DEVICE IS UNNECESSARY
	p.pow[2]->syncHost();
	p.pow[4]->syncHost();
	p.pow[6]->syncHost();
	p.pow[8]->syncHost();
	p.pow[10]->syncHost();
	////

	// Find mVal
	d4 = std::pow(p.pow[4]->getNorm(1), (1.0 / 4));
	d6 = std::pow(p.pow[6]->getNorm(1), (1.0 / 6));
	eta1 = utils::max(d4, d6);
	if ((eta1 <= theta[0]) && (ell(A, coef[0], 3) == 0)) {
		p.mVal = 3;
		return p;
	}
	if ((eta1 <= theta[1]) && (ell(A, coef[1], 5) == 0)) {
		p.mVal = 5;
		return p;
	}
	if (true) { //(A.isSmall()) {
		d8 = std::pow(p.pow[8]->getNorm(1), (1.0 / 8));
	} else {
		//d8 = pow(p.pow[4]->getNormAm(2), (1.0 / 8));
	}
	eta3 = utils::max(d6, d8);
	if ((eta3 <= theta[2]) && (ell(A, coef[2], 7) == 0)) {
		p.mVal = 7;
		return p;
	}
	if ((eta3 <= theta[3]) && (ell(A, coef[3], 9) == 0)) {
		p.mVal = 9;
		return p;
	}
	if (true) { //(A.isSmall()) {
		d10 = std::pow(p.pow[10]->getNorm(1), (1.0 / 10));
	} else {
		//d10 = std::pow(p.pow[2]->getNormAm(5), (1.0 / 10));
	}
	// Find scaling factor
	eta4 = utils::max(d8, d10);
	eta5 = utils::min(eta3, eta4);
	p.scale = utils::max((int) (ceil(log2(eta5 / theta[4]))), 0);
	CUDAMatrix sA(ar, ac);
	double multiplier = 1.0 / std::pow(2, p.scale);
	CUDAMatrix::mul(A, multiplier, sA);
	p.scale += ell(sA, coef[4], 13);
	if (std::isinf((double) p.scale)) {
		std::cout << "S = INF" << std::endl;
		int exp;																		// THIS CODE IS NOT ERROR CHECKED!!!!!
		double t = std::frexp(A.getNorm(1) / theta[4], &exp);
		p.scale = exp - (t == 0.5);
	}
	p.mVal = 13;
	return p;
}

std::vector<double> CUDAMatrix::getPadeCoefficients(int m) {
	switch (m) {
		case 3:
			return { 120, 60, 12, 1 };
		case 5:
			return { 30240, 15120, 3360, 420, 30, 1 };
		case 7:
			return { 17297280, 8648640, 1995840, 277200, 25200, 1512, 56, 1 };
		case 9:
			return { 17643225600, 8821612800, 2075673600, 302702400, 30270240, 2162160, 110880, 3960, 90, 1 };
		case 13:
			return { 64764752532480000, 32382376266240000, 7771770303897600, 1187353796428800, 129060195264000, 10559470521600, 670442572800, 33522128640, 1323241920, 40840800, 960960, 16380, 182, 1 };
		default:
			throw std::runtime_error("Invalid m value");
	}
}

// CONSTRUCTORS

CUDAMatrix::CUDAMatrix() {
	initialised = false;
}

CUDAMatrix::CUDAMatrix(int inNumRowsCols) {
	init(inNumRowsCols, inNumRowsCols);
	setMatrix(0.0);
}

CUDAMatrix::CUDAMatrix(int inNumRows, int inNumCols) {
	init(inNumRows, inNumCols);
	setMatrix(0.0);
}

CUDAMatrix::CUDAMatrix(int inNumRowsCols, std::initializer_list<std::complex<double>> inMatrix) {
	if (inMatrix.size() == inNumRowsCols*inNumRowsCols) {
		init(inNumRowsCols, inNumRowsCols);
		setMatrix(inMatrix);
	} else {
		throw std::runtime_error("Initialiser-list size does not match matrix size");
	}
}

CUDAMatrix::CUDAMatrix(int inNumRows, int inNumCols, std::initializer_list<std::complex<double>> inMatrix) {
	if (inMatrix.size() == inNumRows*inNumCols) {
		init(inNumRows, inNumCols);
		setMatrix(inMatrix);
	} else {
		throw std::runtime_error("Initialiser-list size does not match matrix size");
	}
}

CUDAMatrix::CUDAMatrix(const CUDAMatrix &obj) {
	if (obj.initialised) {
		h_matrix = obj.h_matrix;
		d_matrix = obj.d_matrix;
		numRows = obj.numRows;
		numCols = obj.numCols;
		numEls = obj.numEls;
		size = obj.size;
		initialised = obj.initialised;
	} else {
		throw std::runtime_error("Cannot copy uninitialised matrix");
	}
}

void CUDAMatrix::init(int inNumRows, int inNumCols) {
	numRows = inNumRows;
	numCols = inNumCols;
	numEls = inNumRows*inNumCols;
	size = sizeof(std::complex<double>) * numEls;
	alloc();
	initialised = true;
}

CUDAMatrix::~CUDAMatrix() {
	dealloc();
}
 
// MATRIX OPERATIONS

CUDATimer CUDAMatrix::add(CUDAMatrix& A, CUDAMatrix& B, CUDAMatrix& R) {
	if (A.isInitialised() && B.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int br = B.getNumRows();
		int bc = B.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == br && br == bc && bc == rr && rr == rc) {
			A.syncDevice();
			B.syncDevice();

			cudaParams cp = getCUDAParams(ar, ac);
			CUDATimer t;

			t.start();
			cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, B.d_matrix, R.d_matrix, A.getNumRows());
			t.stop();

			R.syncHost();
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::add(CUDAMatrix& A, std::complex<double> scalar, CUDAMatrix& R) {
	if (A.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == rr && rr == rc) {
			A.syncDevice();

			cudaParams cp = getCUDAParams(ar, ac);
			CUDATimer t;
			
			t.start();
			cudaAddScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, scalar, R.d_matrix, A.getNumRows());
			t.stop();

			R.syncHost();
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::sub(CUDAMatrix& A, CUDAMatrix& B, CUDAMatrix& R) {
	if (A.isInitialised() && B.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int br = B.getNumRows();
		int bc = B.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == br && br == bc && bc == rr && rr == rc) {
			A.syncDevice();
			B.syncDevice();

			cudaParams cp = getCUDAParams(ar, ac);
			CUDATimer t;
			
			t.start();
			cudaSub KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, B.d_matrix, R.d_matrix, A.getNumRows());
			t.stop();

			R.syncHost();
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::sub(CUDAMatrix& A, std::complex<double> scalar, CUDAMatrix& R) {
	if (A.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == rr && rr == rc) {
			A.syncDevice();

			cudaParams cp = getCUDAParams(ar, ac);
			CUDATimer t;

			t.start();
			cudaSubScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, scalar, R.d_matrix, A.getNumRows());
			t.stop();

			R.syncHost();
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::mul(CUDAMatrix& A, CUDAMatrix& B, CUDAMatrix& R) {
	if (A.isInitialised() && B.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int br = B.getNumRows();
		int bc = B.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == br && br == bc && bc == rr && rr == rc) {
			A.syncDevice();
			B.syncDevice();

			cudaParams cp = getCUDAParams(ar, ac);
			CUDATimer t;
			
			t.start();
			cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, B.d_matrix, R.d_matrix, A.getNumRows());
			t.stop();

			R.syncHost();
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::mul(CUDAMatrix& A, std::complex<double> scalar, CUDAMatrix& R) {
	if (A.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == rr && rr == rc) {
			A.syncDevice();

			cudaParams cp = getCUDAParams(ar, ac);
			CUDATimer t;
			
			t.start();
			cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, scalar, R.d_matrix, A.getNumRows());
			t.stop();

			R.syncHost();
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::pow(CUDAMatrix& A, int pow, CUDAMatrix& R) {
	if (A.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == rr && rr == rc) {
			A.syncDevice();
			CUDAMatrix T(ar);
			T.setIdentity();
			T.syncDevice();

			cudaParams cp = getCUDAParams(ar, ac);
			CUDATimer t;

			t.start();
			for (int c1 = 0; c1 < pow; c1++) {
				cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, T.d_matrix, T.d_matrix, ar);
			}
			t.stop();

			T.syncHost();
			R.setMatrix(T.getMatrix());
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::inv(CUDAMatrix& A, CUDAMatrix& R) {
	if (A.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == rr && rr == rc) {
			
			CUDATimer t;
			CUDAMatrix L = CUDAMatrix(ar, ac);
			CUDAMatrix U = CUDAMatrix(ar, ac);
			CUDAMatrix Z = CUDAMatrix(ar, ac);
			CUDAMatrix I = CUDAMatrix(ar, ac);
			I.setIdentity();

			t.start();

			int n = ar;
			int i, j, k;
			// LU Decomposition
			for (i = 0; i < n; i++) {
				for (j = 0; j < n; j++) {
					if (j < i) {
						U.setCell(i, j, 0);
					} else {
						U.setCell(i, j, A.getCell(i, j));
						for (k = 0; k < i; k++) {
							U.setCell(i, j, (U.getCell(i, j) - U.getCell(k, j) * L.getCell(i, k)));
						}
					}
				}
				for (j = 0; j < n; j++) {
					if (j < i) {
						L.setCell(j, i, 0);
					} else if (j == i) {
						L.setCell(j, i, 1);
					} else {
						L.setCell(j, i, (A.getCell(j, i) / U.getCell(i, i)));
						for (k = 0; k < i; k++) {
							L.setCell(j, i, (L.getCell(j, i) - ((U.getCell(k, i) * L.getCell(j, k)) / U.getCell(i, i))));
						}
					}
				}
			}
			for (i = 0; i < n; i++) {
				// Find Z (L^-1) with Forward Substitution
				for (j = 0; j < n; j++) {
					Z.setCell(j, i, I.getCell(j, i));
					for (k = 0; k < n; k++) {
						if (k != j) {
							Z.setCell(j, i, (Z.getCell(j, i) - (L.getCell(j, k) * Z.getCell(k, i))));
						}
					}
				}
				// Find X (A^-1) with Backward Substitution
				for (j = n - 1; j >= 0; j--) {
					R.setCell(j, i, Z.getCell(j, i));
					for (k = 0; k < n; k++) {
						if (k != j) {
							R.setCell(j, i, (R.getCell(j, i) - (U.getCell(j, k) * R.getCell(k, i))));
						}
					}
					R.setCell(j, i, R.getCell(j, i) / U.getCell(j, j));
				}
			}

			t.stop();
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::tra(CUDAMatrix& A, CUDAMatrix& R) {
	if (A.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ac == rr) {
			A.syncDevice();

			int c1, c2;
			CUDATimer t;
			
			t.start();
			for (c1 = 0; c1 < A.getNumRows(); c1++) {
				for (c2 = 0; c2 < A.getNumCols(); c2++) {
					R.setCell(c1, c2, A.getCell(c2, c1));
				}
			}
			t.stop();

			R.syncDevice();
			return t;
		} else {
			throw std::runtime_error("Transpose matrix is the wrong size");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::exp(CUDAMatrix& A, CUDAMatrix& R) {
	if (A.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == rr && rr == rc) {
			A.syncDevice();
			CUDATimer t;
			int c1, c2;
			int n = utils::max(ar, ac);
			// Special Cases
			if (A.isDiagonal()) {
				t.start();
				for (c1 = 0; c1 < n; c1++) {
					R.setCell(c1, c1, std::exp(A.getCell(c1, c1)));
				}
				t.stop();
				R.syncDevice();
			} else if (A.isZero()) {
				t.start();
				R.setMatrix(0);
				t.stop();
				R.syncDevice();
			// Normal Case
			} else {
				// Create Matrices
				CUDAMatrix U(ar, ac);
				CUDAMatrix V(ar, ac);
				CUDAMatrix I(ar, ac); // Identity
				CUDAMatrix T(ar, ac); // Tally
				CUDAMatrix TMP(ar, ac); // Temporary
				I.setIdentity();
				I.syncDevice();
				// Get CUDA params
				cudaParams cp = getCUDAParams(ar, ac);
				// Get Pade params
				padeParams p = getPadeParams(A);
				int s = p.scale;
				int m = p.mVal;
				std::vector<CUDAMatrix*> pow = p.pow;
				// Get Pade coefficients
				std::vector<double> c = getPadeCoefficients(m);
				// Start timer
				t.start();
				// Scaling
				if (s != 0) {
					double multiplier;
					multiplier = 1.0 / std::pow(2, s);
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, multiplier, A.d_matrix, n);
					for (c1 = 2; c1 <= 6; c1 += 2) {
						multiplier = 1.0 / std::pow(2, (s * c1));
						cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[c1]->d_matrix, multiplier, pow[c1]->d_matrix, n);
					}
				}
				// Approximation
				if (m == 3 || m == 5 || m == 7 || m == 9) {
					for (c1 = (int) (pow.size()) + 2; c1 < m - 1; c1 += 2) { //for (k = strt:2:m-1)
						cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[c1 - 2]->d_matrix, pow[2]->d_matrix, pow[c1]->d_matrix, n);
					}
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (I.d_matrix, c[1], U.d_matrix, n);
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (I.d_matrix, c[0], V.d_matrix, n);
					for (c2 = m; c2 >= 3; c2 -= 2) { //for (j = m : -2 : 3)
						cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[c2 - 1]->d_matrix, c[c2], TMP.d_matrix, n);
						cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (U.d_matrix, TMP.d_matrix, U.d_matrix, n);
						cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[c2 - 1]->d_matrix, c[c2-1], TMP.d_matrix, n);
						cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (V.d_matrix, TMP.d_matrix, V.d_matrix, n);
					}
					cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (U.d_matrix, A.d_matrix, U.d_matrix, n);
				} else if (m == 13) {
					// This is the equivellent of .. 
					// U = A * (p[6] * (c[13] * p[6] + c[11] * p[4] + c[9] * p[2]) + c[7] * p[6] + c[5] * p[4] + c[3] * p[2] + c[1] * I);		RUN IN STREAM 1
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[6]->d_matrix, c[13], T.d_matrix, n);		// p[6] * c[13] -> T			Needs new TMP var
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[4]->d_matrix, c[11], TMP.d_matrix, n);		// p[4] * c[11] -> TMP			(Cannot be used in multiple streams)
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[2]->d_matrix, c[9], TMP.d_matrix, n);		// p[2] * c[9]  -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[6]->d_matrix, T.d_matrix, T.d_matrix, n);			// p[6] * T     -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[6]->d_matrix, c[7], TMP.d_matrix, n);		// p[6] * c[7]  -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[4]->d_matrix, c[5], TMP.d_matrix, n);		// p[4] * c[5]  -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[2]->d_matrix, c[3], TMP.d_matrix, n);		// p[2] * c[3]  -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (I.d_matrix, c[1], TMP.d_matrix, n);				// I * c[1]     -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, T.d_matrix, U.d_matrix, n);				// A * T        -> U
					// This is the equivellent of ..
					//V = p[6] * (c[12] * p[6] + c[10] * p[4] + c[8] * p[2]) + c[6] * p[6] + c[4] * p[4] + c[2] * p[2] + c[0] * I;				RUN IN STREAM 2
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[6]->d_matrix, c[12], T.d_matrix, n);		// p[6] * c[12] -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[4]->d_matrix, c[10], TMP.d_matrix, n);		// p[4] * c[10] -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[2]->d_matrix, c[8], TMP.d_matrix, n);		// p[2] * c[8]  -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[6]->d_matrix, T.d_matrix, T.d_matrix, n);			// p[6]			-> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[6]->d_matrix, c[6], TMP.d_matrix, n);		// p[6] * c[6]  -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[4]->d_matrix, c[4], TMP.d_matrix, n);		// p[4] * c[4]  -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (pow[2]->d_matrix, c[2], TMP.d_matrix, n);		// p[2] * c[2]  -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);				// T + TMP      -> T
					cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (I.d_matrix, c[0], TMP.d_matrix, n);				// I * c[0]     -> TMP
					cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, V.d_matrix, n);				// T + TMP      -> V
				}
				// This is the equivellent of ..
				// R = (V - U) / (2 * U) + I;  ||?? R = (-U + V) / (U + V);
				cudaSub KERNEL_ARGS2(cp.bpg, cp.tpb) (V.d_matrix, U.d_matrix, T.d_matrix, n);
				cudaMulScalar KERNEL_ARGS2(cp.bpg, cp.tpb) (U.d_matrix, 2, TMP.d_matrix, n);
				//cudaInv KERNEL_ARGS2(cp.bpg, cp.tpb) (TMP.d_matrix, TMP.d_matrix, n); // TEMP CODE BELOW
				T.syncHost();
				CUDAMatrix::inv(T, T);
				T.syncDevice();
				//
				cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, TMP.d_matrix, T.d_matrix, n);
				cudaAdd KERNEL_ARGS2(cp.bpg, cp.tpb) (T.d_matrix, I.d_matrix, R.d_matrix, n);
				// Squaring
				for (int k = 0; k < s; k++) {
					cudaMul KERNEL_ARGS2(cp.bpg, cp.tpb) (R.d_matrix, R.d_matrix, R.d_matrix, n);
				}
				hipDeviceSynchronize();
				t.stop();
				R.syncHost();
			}
			return t;
		} else {
			throw std::runtime_error("Matrix sizez do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

CUDATimer CUDAMatrix::abs(CUDAMatrix& A, CUDAMatrix& R) {
	if (A.isInitialised() && R.isInitialised()) {
		int ar = A.getNumRows();
		int ac = A.getNumCols();
		int rr = R.getNumRows();
		int rc = R.getNumCols();
		if (ar == ac && ac == rr && rr == rc) {
			A.syncDevice();

			cudaParams cp = getCUDAParams(ar, ac);
			CUDATimer t;

			t.start();
			cudaAbs KERNEL_ARGS2(cp.bpg, cp.tpb) (A.d_matrix, R.d_matrix, ar);
			t.stop();

			R.syncHost();
			return t;
		} else {
			throw std::runtime_error("Matrix sizes do not match");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

// BOOLEANS

bool CUDAMatrix::isInitialised() {
	return initialised;
}

bool CUDAMatrix::isSquare() {
	if (initialised) {
		if (numCols == numRows) {
			return true;
		} else {
			return false;
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

bool CUDAMatrix::isDiagonal() {
	if (initialised) {
		if (!isSquare()) {
			return false;
		}
		for (int c1 = 0; c1 < numRows; c1++) {
			for (int c2 = 0; c2 < numCols; c2++) {
				if (c1 != c2 && getCell(c1, c2) != 0.0) {
					return false;
				}
			}
		}
		return true;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

bool CUDAMatrix::isIdentity() {
	if (initialised) {
		for (int c1 = 0; c1 < numRows; c1++) {
			for (int c2 = 0; c2 < numCols; c2++) {
				if ((c1 != c2 && getCell(c1, c2) != 0.0) || (c1 == c2 && getCell(c1, c2) != 1.0)) {
					return false;
				}
			}
		}
		return true;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

bool CUDAMatrix::isZero() {
	if (initialised) {
		for (int c1 = 0; c1 < numRows; c1++) {
			for (int c2 = 0; c2 < numCols; c2++) {
				if (getCell(c1, c2) != 0.0) {
					return false;
				}
			}
		}
		return true;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

bool CUDAMatrix::isSmall() {
	return utils::max(numRows, numCols) < 150;
}

bool CUDAMatrix::isComplex() {
	std::complex<double> cell;
	for (int c1 = 0; c1 < numEls; c1++) {
		cell = getCell(c1);
		if (cell.imag() != 0.0) {
			return true;
		}
	}
	return false;
}

// SETTERS

void CUDAMatrix::setCell(int row, int col, std::complex<double> val) {
	if (isInitialised()) {
		h_matrix[numCols * row + col] = val;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

void CUDAMatrix::setCell(int i, std::complex<double> val) {
	if (isInitialised()) {
		h_matrix[i] = val;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

void CUDAMatrix::setMatrix(std::complex<double> val) {
	if (isInitialised()) {
		for (int c1 = 0; c1 < getNumEls(); c1++) {
			h_matrix[c1] = val;
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

void CUDAMatrix::setMatrix(std::complex<double>* inMatrix) {
	if (isInitialised()) {
		for (int c1 = 0; c1 < numEls; c1++) {
			h_matrix[c1] = inMatrix[c1];
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

void CUDAMatrix::setMatrix(std::initializer_list<std::complex<double>> inMatrix) {
	if (isInitialised()) {
		if (inMatrix.size() == getNumEls()) {
			std::copy(inMatrix.begin(), inMatrix.end(), h_matrix);
		} else {
			throw std::runtime_error("Initialiser-list size does not match matrix size");
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

void CUDAMatrix::setIdentity() {
	if (isInitialised()) {
		int row, col;
		for (int c1 = 0; c1 < getNumEls(); c1++) {
			row = getCurRow(c1);
			col = getCurCol(c1);
			if (row == col) {
				h_matrix[c1] = 1;
			} else {
				h_matrix[c1] = 0;
			}
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

void CUDAMatrix::setRandomDouble(double min, double max) {
	if (isInitialised()) {
		double r;
		std::default_random_engine rng((unsigned int) (time(0)));
		std::uniform_real_distribution<double> gen(min, max);
		for (int c1 = 0; c1 < numEls; c1++) {
			r = gen(rng);
			setCell(c1, r);
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

void CUDAMatrix::setRandomInt(int min, int max) {
	if (isInitialised()) {
		int r;
		std::default_random_engine rng((unsigned int) (time(0)));
		std::uniform_int_distribution<int> gen(min, max);
		for (int c1 = 0; c1 < numEls; c1++) {
			r = gen(rng);
			setCell(c1, r);
		}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

// GETTERS

double CUDAMatrix::getNorm(int n) {
	int c1, c2;
	double sum, max = 0;
	if (n == 1) {
		// 1 Norm
		for (c1 = 0; c1 < numCols; c1++) {
			sum = 0;
			for (c2 = 0; c2 < numRows; c2++) {
				sum += std::abs(getCell(c2, c1));
			}
			if (std::norm(sum) > std::norm(max)) {
				max = sum;
			}
		}
		return max;
	} else if (n == INFINITY) {
		// Inf Norm
		for (c1 = 0; c1 < numRows; c1++) {
			sum = 0;
			for (c2 = 0; c2 < numCols; c2++) {
				sum += std::abs(getCell(c2, c1));
			}
			if (std::norm(sum) > std::norm(max)) {
				max = sum;
			}
		}
		return max;
	} else {
		//// Euclidian									Not called from anywhere. Requires SVD implementation to work.
		//sum = 0;
		//for (c1 = 0; c1 < numEls; c1++) {
		//	sum += std::pow(getCell(c1), n);
		//}
		//return std::pow(sum, 1.0 / n);
		return -1;
	}
}

int CUDAMatrix::getCurRow(int i) {
	if (isInitialised()) {
		return (int) (floor(i / numCols));
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

int CUDAMatrix::getCurCol(int i) {
	if (isInitialised()) {
		return (int) (i - (numCols*getCurRow(i)));
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

std::complex<double> CUDAMatrix::getCell(int row, int col) {
	if (isInitialised()) {
		return h_matrix[row*numCols + col];
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

std::complex<double> CUDAMatrix::getCell(int i) {
	if (isInitialised()) {
		return h_matrix[i];
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

std::complex<double>* CUDAMatrix::getMatrix() {
	if (isInitialised()) {
		return h_matrix;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

int CUDAMatrix::getNumRows() {
	if (isInitialised()) {
		return numRows;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

int CUDAMatrix::getNumCols() {
	if (isInitialised()) {
		return numCols;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

int CUDAMatrix::getNumEls() {
	if (isInitialised()) {
		return numEls;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

size_t CUDAMatrix::getSize() {
	if (isInitialised()) {
		return size;
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}

// UTILS

int utils::getNumDigits(double x) {
	if (x > 1.0 || x < -1.0) {
		return (int) (floor(log10(abs(x))) + 1);
	}
	return 1;
}

int utils::max(int x, int y) {
	if (x > y) {
		return x;
	} else {
		return y;
	}
}

double utils::max(double x, double y) {
	if (x > y) {
		return x;
	} else {
		return y;
	}
}

int utils::min(int x, int y) {
	if (x < y) {
		return x;
	} else {
		return y;
	}
}

double utils::min(double x, double y) {
	if (x < y) {
		return x;
	} else {
		return y;
	}
}

// OPERATOR OVERRIDES


std::ostream& operator<<(std::ostream& oStream, CUDAMatrix& A) {
	if (A.isInitialised()) {
		// Init
		std::complex<double> cell;
		bool isComplex = A.isComplex();
		bool scientific = false;
		int c1, c2, r, i;
		int realLength = 0, imagLength = 0, exp = 0;
		double divider;
		int precision = 0;
		int maxFixedDigits = 4;
		// Get info
		for (c1 = 0; c1 < A.getNumEls(); c1++) {
			cell = A.getCell(c1);
			// Check if it's decimal
			if ((cell.real() - (int) (cell.real())) != 0.0 ||
				(cell.imag() - (int) (cell.imag())) != 0.0) {
				precision = 4;
			}
			// Get maximum exponent
			r = utils::getNumDigits(cell.real());
			i = utils::getNumDigits(cell.imag());
			if (r - 1 > exp) {
				exp = r - 1;
				realLength = r;
			}
			if (i - 1 > exp) {
				exp = i - 1;
				imagLength = i;
				if (abs(cell.imag() == 1.0)) {
					imagLength++;
				}
			}
		}
		// Check if the output should be in fixed or scientific form
		if (exp >= maxFixedDigits) {
			scientific = true;
		}
		// Get divider for scientific form
		divider = std::pow(10, exp);

		// Output name and multiplier
		oStream << " = ";
		if (scientific) {
			oStream << "(10 ^ " << exp << ") *";
		}
		// Output cell
		oStream << std::endl << std::setprecision(precision) << std::fixed;
		for (c1 = 0; c1 < A.getNumEls(); c1++) {
			cell = A.getCell(c1);
			oStream << "| ";
			// Spacing and formatting for scientific/fixed
			if (scientific) {
				cell /= divider;
			} else {
				r = utils::getNumDigits(cell.real());
				for (c2 = 0; c2 < (realLength - r); c2++) {
					oStream << " ";
				}
			}
			// Output real
			oStream << cell.real() << " ";
			// Output complex
			if (isComplex) {
				if (cell.imag() != 0.0) {
					if (cell.imag() > 0.0) {
						oStream << "+ ";
					} else {
						oStream << "- ";
					}
					if (abs(cell.imag()) != 1.0) {
						oStream << std::abs(cell.imag());
					} else {
						oStream << " ";
					}
					oStream << "i ";
				} else {
					i = utils::getNumDigits(cell.imag());
					for (c2 = 0; c2 < imagLength + 3; c2++) {
						oStream << " ";
					}
				}
			}
			// Output new line if row end reached
			if (A.getCurRow(c1 + 1) > A.getCurRow(c1)) {
				oStream << "|";
				if (A.getCurRow(c1 + 1) < A.getNumRows()) {
					oStream << std::endl;
				}
			}
		}
		oStream << std::endl;
		return oStream;
		//	// Get precision
		//	cell = A.getCell(c1);
		//	if ((cell - (int) (cell)) != 0.0) {
		//		precision = 5;
		//	}
		//	// Get maximum number length
		//	length = utils::getNumDigits(cell);
		//	if (length > maxLength) {
		//		maxLength = length;
		//	}
		//}
		//for (c1 = 0; c1 < A.getNumEls(); c1++) {
		//	cell = A.getCell(c1);
		//	// Remove negative zeros
		//	if (cell == 0.0) {
		//		cell = 0;
		//	}
		//	oStream << "| ";
		//	// Add whitespace if shorter than maxLength
		//	length = utils::getNumDigits(cell);
		//	for (c2 = 0; c2 < (maxLength - length); c2++) {
		//		oStream << " ";
		//	}
		//	// Output number
		//	oStream << std::setprecision(precision) << std::fixed << cell << " ";
		//	// Output new line if row end reached
		//	if (A.getCurRow(c1 + 1) > A.getCurRow(c1)) {
		//		oStream << "|";
		//		if (A.getCurRow(c1 + 1) < A.getNumRows()) {
		//			oStream << std::endl;
		//		}
		//	}
	} else {
		throw std::runtime_error("Cannot perform matrix operations before initialisation");
	}
}