//
// Cardiff University | Computer Science
// Module:     CM3203 One Semester Project (40 Credits)
// Title:      Parallelisation of Matrix Exponentials in C++/CUDA for Quantum Control
// Date:       2016
//
// Author:     Peter Davison
// Supervisor: Dr. Frank C Langbein
// Moderator:  Dr. Irena Spasic
//

// Include header file
#include "CUDATimer.cuh"

void CUDATimer::start() {
	clear();
	hipEventCreate(&t1);
	hipEventCreate(&t2);
	hipEventRecord(t1, 0);
}

void CUDATimer::stop() {
	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);
	hipEventElapsedTime(&time, t1, t2);
	hipEventDestroy(t1);
	hipEventDestroy(t2);
}

void CUDATimer::clear() {
	time = 0;
}

float CUDATimer::getTime() {
	return time;
}

std::ostream& operator<<(std::ostream& oStream, CUDATimer& t) {
	oStream << std::setprecision(10) << std::fixed << t.getTime()/1000 << "s" << std::endl;
	return oStream;
}